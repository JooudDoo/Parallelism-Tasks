#include "hip/hip_runtime.h"
#include <iostream>

#include <hipcub/hipcub.hpp>
#include <cub/block/block_reduce.cuh>

#include "hip/hip_runtime.h"

#include "sub.cuh" // contains functions for processing arguments and displaying them

#ifdef NVPROF_
#include </opt/nvidia/hpc_sdk/Linux_x86_64/22.11/cuda/11.8/targets/x86_64-linux/include/nvtx3/nvToolsExt.h>
#endif

#define at(arr, x, y) (arr[(x) * (n) + (y)])

// Values
constexpr int MAXIMUM_THREADS_PER_BLOCK = 32;
constexpr int THREADS_PER_BLOCK_REDUCE = 256;

// Cornerns
constexpr int LEFT_UP = 10;
constexpr int LEFT_DOWN = 20;
constexpr int RIGHT_UP = 20;
constexpr int RIGHT_DOWN = 30;

// Other values
constexpr int ITERS_BETWEEN_UPDATE = 400;

// Function definitions
void initArrays(double *mainArr, double *main_D, double *sub_D, cmdArgs *args);

__global__ void iterate(double *F, double *Fnew, const cmdArgs *args);

__global__ void block_reduce(const double *in1, const double *in2, const int n, double *out);

int main(int argc, char *argv[]){
    hipSetDevice(2); // selecting free GPU device
    cmdArgs args = cmdArgs{false, false, 1E-6, (int)1E6, 10, 10}; // create default command line arguments
    processArgs(argc, argv, &args);
    printSettings(&args);

    double *F_H;
    double *F_D, *Fnew_D;
    size_t size = args.n * args.m * sizeof(double);
    double error = 0;
    int iterationsElapsed = 0;

    hipMalloc(&F_D, size);
    hipMalloc(&Fnew_D, size);

    F_H = (double *)calloc(sizeof(double), size);

    initArrays(F_H, F_D, Fnew_D, &args);


    {
        size_t grid_size = args.n * args.m;

        cmdArgs *args_d;
        hipMalloc(&args_d, sizeof(cmdArgs));
        hipMemcpy(args_d, &args, sizeof(cmdArgs), hipMemcpyHostToDevice);

        int num_blocks_reduce = (grid_size + THREADS_PER_BLOCK_REDUCE - 1) / THREADS_PER_BLOCK_REDUCE;

        double *error_reduction;
        hipMalloc(&error_reduction, sizeof(double) * num_blocks_reduce);
        double *error_d;
        hipMalloc(&error_d, sizeof(double));

        // prepare graph
        hipStream_t stream;
        hipStreamCreate(&stream);
        hipGraph_t graph;
        hipGraphExec_t graph_instance;

        // prepare reduction
        void *d_temp_storage = nullptr;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, error_reduction, error_d, num_blocks_reduce, stream);
        hipMalloc(&d_temp_storage, temp_storage_bytes);

        hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, error_reduction, error_d, 1024, stream);

        hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

        dim3 threadPerBlock = dim3((args.n + MAXIMUM_THREADS_PER_BLOCK - 1) / MAXIMUM_THREADS_PER_BLOCK, (args.m + MAXIMUM_THREADS_PER_BLOCK - 1) / MAXIMUM_THREADS_PER_BLOCK);
        dim3 blocksPerGrid = dim3((args.n + ((args.m + MAXIMUM_THREADS_PER_BLOCK - 1) / MAXIMUM_THREADS_PER_BLOCK) - 1) / ((args.m + MAXIMUM_THREADS_PER_BLOCK - 1) / MAXIMUM_THREADS_PER_BLOCK),
            (args.n + ((args.n + MAXIMUM_THREADS_PER_BLOCK - 1) / MAXIMUM_THREADS_PER_BLOCK) - 1) / ((args.m + MAXIMUM_THREADS_PER_BLOCK - 1) / MAXIMUM_THREADS_PER_BLOCK));

        for (size_t i = 0; i < ITERS_BETWEEN_UPDATE / 2; i++) {
            iterate<<<blocksPerGrid, threadPerBlock, 0, stream>>>(F_D, Fnew_D, args_d);
            iterate<<<blocksPerGrid, threadPerBlock, 0, stream>>>(Fnew_D, F_D, args_d);
        }

        hipStreamEndCapture(stream, &graph);
        hipGraphInstantiate(&graph_instance, graph, NULL, NULL, 0);
#ifdef NVPROF_
    nvtxRangePush("MainCycle");
#endif
        do {

            hipGraphLaunch(graph_instance, stream);
            hipDeviceSynchronize();

            block_reduce<<<num_blocks_reduce, THREADS_PER_BLOCK_REDUCE>>>(F_D, Fnew_D, grid_size, error_reduction);
            hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, error_reduction, error_d, num_blocks_reduce, stream);

            hipMemcpy(&error, error_d, sizeof(double), hipMemcpyDeviceToHost);

            iterationsElapsed += ITERS_BETWEEN_UPDATE;
        } while (error > args.eps && iterationsElapsed < args.iterations);
#ifdef NVPROF_
    nvtxRangePop();
#endif
        hipGraphDestroy(graph);
        hipStreamDestroy(stream);
        
    }


    std::cout << "Iterations: " << iterationsElapsed << std::endl;
    std::cout << "Error: " << error << std::endl;
    if (args.showResultArr) {
    hipMemcpy(F_H, Fnew_D, size, hipMemcpyDeviceToHost);
    int n = args.n;
    for (int x = 0; x < args.n; x++) {
        for (int y = 0; y < args.m; y++) {
            std::cout << at(F_H, x, y) << ' ';
        }
        std::cout << std::endl;
    }
    }

    hipFree(F_D);
    hipFree(Fnew_D);
    free(F_H);
    return 0;
}

void initArrays(double *mainArr, double *main_D, double *sub_D, cmdArgs *args){
    int n = args->n;
    int m = args->m;
    size_t size = n * m * sizeof(double);

    for (int i = 0; i < n * m && args->initUsingMean; i++)
    {
    mainArr[i] = (LEFT_UP + LEFT_DOWN + RIGHT_UP + RIGHT_DOWN) / 4;
    }

    at(mainArr, 0, 0) = LEFT_UP;
    at(mainArr, 0, m - 1) = RIGHT_UP;
    at(mainArr, n - 1, 0) = LEFT_DOWN;
    at(mainArr, n - 1, m - 1) = RIGHT_DOWN;
    for (int i = 1; i < n - 1; i++)
    {
    at(mainArr, 0, i) = (at(mainArr, 0, m - 1) - at(mainArr, 0, 0)) / (m - 1) * i + at(mainArr, 0, 0);
    at(mainArr, i, 0) = (at(mainArr, n - 1, 0) - at(mainArr, 0, 0)) / (n - 1) * i + at(mainArr, 0, 0);
    at(mainArr, n - 1, i) = (at(mainArr, n - 1, m - 1) - at(mainArr, n - 1, 0)) / (m - 1) * i + at(mainArr, n - 1, 0);
    at(mainArr, i, m - 1) = (at(mainArr, n - 1, m - 1) - at(mainArr, 0, m - 1)) / (m - 1) * i + at(mainArr, 0, m - 1);
    }
    hipMemcpy(main_D, mainArr, size, hipMemcpyHostToDevice);
    hipMemcpy(sub_D, mainArr, size, hipMemcpyHostToDevice);
}

__global__ void iterate(double *F, double *Fnew, const cmdArgs *args){

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (j == 0 || i == 0 || i == args->n - 1 || j == args->n - 1) return; // Don't update borders

    int n = args->n;
    at(Fnew, i, j) = 0.25 * (at(F, i + 1, j) + at(F, i - 1, j) + at(F, i, j + 1) + at(F, i, j - 1));
}

__global__ void block_reduce(const double *in1, const double *in2, const int n, double *out){
    typedef hipcub::BlockReduce<double, 256> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    double max_diff = 0;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x)
    {
    double diff = abs(in1[i] - in2[i]);
    max_diff = fmax(diff, max_diff);
    }

    double block_max_diff = BlockReduce(temp_storage).Reduce(max_diff, hipcub::Max());

    if (threadIdx.x == 0)
    {
    out[blockIdx.x] = block_max_diff;
    }
}