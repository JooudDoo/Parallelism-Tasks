#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void HelloWorld()
{
    printf("Hello world, %d, %d\n", blockIdx.x, threadIdx.x);
}

int main()
{
    HelloWorld <<<2, 5 >>>();
    hipDeviceSynchronize();
    return 0;
}