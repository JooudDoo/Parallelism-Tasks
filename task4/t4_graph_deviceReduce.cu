#include "hip/hip_runtime.h"
#include <iostream>

#include <hipcub/hipcub.hpp>
#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>
#include <cub/block/block_reduce.cuh>

#include "hip/hip_runtime.h"

#include "sub.cuh" // contains functions for processing arguments and displaying them

#ifdef NVPROF_
#include </opt/nvidia/hpc_sdk/Linux_x86_64/22.11/cuda/11.8/targets/x86_64-linux/include/nvtx3/nvToolsExt.h>
#endif

#define at(arr, x, y) (arr[(x)*(n)+(y)])

constexpr int MAXIMUM_THREADS_PER_BLOCK = 32;

// Cornerns
constexpr int LEFT_UP = 10;
constexpr int LEFT_DOWN = 20;
constexpr int RIGHT_UP = 20;
constexpr int RIGHT_DOWN = 30;

constexpr int ITERS_BETWEEN_UPDATE = 400;

void initArrays(double* mainArr, double* main_D, double* sub_D, cmdArgs* args);

__global__ void iterate(double* F, double* Fnew, double* subs, const cmdArgs* args);

int main(int argc, char *argv[]){
    hipSetDevice(2);
    cmdArgs args = cmdArgs{false, false, 1E-6, (int)1E6, 10, 10}; // create default command line arguments 
    processArgs(argc, argv, &args);
    printSettings(&args);

    double* F_H;
    double* F_D, *Fnew_D;
    size_t size = args.n*args.m*sizeof(double);
    double error = 0;
    int iterationsElapsed = 0;

    hipMalloc(&F_D, size);
    hipMalloc(&Fnew_D, size);
    
    F_H = (double*)calloc(sizeof(double), size);

    initArrays(F_H, F_D, Fnew_D, &args);

    // Основной алгоритм здесь


    {
    size_t grid_size = args.n * args.m;

    cmdArgs* args_d;
    hipMalloc(&args_d, sizeof(cmdArgs));
    hipMemcpy(args_d, &args, sizeof(cmdArgs), hipMemcpyHostToDevice);

    double* substractions;
    hipMalloc(&substractions, size);

    double* error_d;
    hipMalloc(&error_d, sizeof(double));

    // prepare graph
    hipStream_t stream;
    hipStreamCreate(&stream);
    hipGraph_t graph;
    hipGraphExec_t instance;

    //prepare reduction
    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, substractions, error_d, grid_size);
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

    dim3 threadPerBlock = dim3((args.n + MAXIMUM_THREADS_PER_BLOCK - 1) / MAXIMUM_THREADS_PER_BLOCK, (args.m + MAXIMUM_THREADS_PER_BLOCK - 1) / MAXIMUM_THREADS_PER_BLOCK);
    dim3 blocksPerGrid = dim3((args.n + ((args.m + MAXIMUM_THREADS_PER_BLOCK - 1) / MAXIMUM_THREADS_PER_BLOCK) - 1) / ((args.m + MAXIMUM_THREADS_PER_BLOCK - 1) / MAXIMUM_THREADS_PER_BLOCK),
            (args.n + ((args.n + MAXIMUM_THREADS_PER_BLOCK - 1) / MAXIMUM_THREADS_PER_BLOCK) - 1) / ((args.m + MAXIMUM_THREADS_PER_BLOCK - 1) / MAXIMUM_THREADS_PER_BLOCK));
    
    for(size_t i = 0; i < ITERS_BETWEEN_UPDATE/2; i++){
        iterate<<<blocksPerGrid, threadPerBlock, 0, stream>>>(F_D, Fnew_D, substractions, args_d);
        iterate<<<blocksPerGrid, threadPerBlock, 0, stream>>>(Fnew_D, F_D, substractions, args_d);
    }
    
    hipStreamEndCapture(stream, &graph);
    hipGraphInstantiate(&instance, graph, NULL, NULL, 0);

#ifdef NVPROF_
    nvtxRangePush("MainCycle");
#endif

    do {

        hipGraphLaunch(instance, stream);
        hipDeviceSynchronize();
        hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, substractions, error_d, grid_size, stream);
        hipMemcpy(&error, error_d, sizeof(double), hipMemcpyDeviceToHost);

        iterationsElapsed += ITERS_BETWEEN_UPDATE;
    } while(error > args.eps && iterationsElapsed < args.iterations);

#ifdef NVPROF_
    nvtxRangePop();
#endif

    hipFree(substractions);
    }


// ----------------------

    std::cout << "Iterations: " << iterationsElapsed << std::endl;
    std::cout << "Error: " << error << std::endl;
    if(args.showResultArr){
        hipMemcpy(F_H, Fnew_D, size, hipMemcpyDeviceToHost);
        int n = args.n;
        for(int x = 0; x < args.n; x++){
            for(int y = 0; y < args.m; y++){ 
                std::cout << at(F_H, x, y) << ' ';
            }
            std::cout << std::endl;
        }
    }

    hipFree(F_D);
    hipFree(Fnew_D);
    free(F_H);
    return 0;
}

void initArrays(double* mainArr, double* main_D, double* sub_D, cmdArgs* args){
    int n = args->n;
    int m = args->m;
    size_t size = n*m*sizeof(double);

    for(int i = 0; i < n*m && args->initUsingMean; i++){
        mainArr[i] = (LEFT_UP+LEFT_DOWN+RIGHT_UP+RIGHT_DOWN)/4;
    }

    at(mainArr, 0, 0) = LEFT_UP;
    at(mainArr, 0, m-1) = RIGHT_UP;
    at(mainArr, n-1, 0) = LEFT_DOWN;
    at(mainArr, n-1, m-1) = RIGHT_DOWN;
    for(int i = 1; i < n-1; i++){
        at(mainArr,0,i) = (at(mainArr,0,m-1)-at(mainArr,0,0))/(m-1)*i+at(mainArr,0,0);
        at(mainArr,i,0) = (at(mainArr,n-1,0)-at(mainArr,0,0))/(n-1)*i+at(mainArr,0,0);
        at(mainArr,n-1,i) = (at(mainArr,n-1,m-1)-at(mainArr,n-1,0))/(m-1)*i+at(mainArr,n-1,0);
        at(mainArr,i,m-1) = (at(mainArr,n-1,m-1)-at(mainArr,0,m-1))/(m-1)*i+at(mainArr,0,m-1);
    }
    hipMemcpy(main_D, mainArr, size, hipMemcpyHostToDevice);
    hipMemcpy(sub_D, mainArr, size, hipMemcpyHostToDevice);
}

__global__ void iterate(double* F, double* Fnew, double* subs, const cmdArgs* args){

    int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

    if(j == 0 || i == 0 || i == args->n-1 || j == args->n-1) return; // Dont update borders

    int n = args->n;
    at(Fnew, i, j) = 0.25 * (at(F, i+1, j) + at(F, i-1, j) + at(F, i, j+1) + at(F, i, j-1));
    at(subs, i, j) = fabs(at(Fnew, i, j) - at(F, i, j));
}
