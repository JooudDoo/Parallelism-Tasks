#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <sstream>
#include <cmath>
#include <ctime>
#include <iomanip>

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#include "mpi.h"

#define UPDATE 250
#define THREADS_MAX 1024
#define THREAD (size < THREADS_MAX ? size : THREADS_MAX)

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Функция изменения матрицы
__global__ void iterate(double* A, double* A_new, size_t size_x, size_t size_y) {
	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    size_t j = blockIdx.y * blockDim.y + threadIdx.y;
	
    if ((i == 0) || (j == 0) || (j == size_y - 1) || (i == size_x - 1)) return; // Don't update borders
    A_new[j * size_x + i] = 0.25 * (A[j * size_x + i - 1] + A[(j - 1) * size_x + i] + A[(j + 1) * size_x + i] + A[j * size_x + i + 1]);	
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Функция разницы матриц
__global__ void subtraction(double* A, double* A_new, double* A_err, size_t size_x) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
	A_err[j * size_x + j] = A[j * size_x + j] - A_new[j * size_x + j];
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Значения по умодчанию
double eps = 1E-6;
int size = 512;
int iter_max = 1E6;

int main(int argc, char** argv) {
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Получение значений из командной строки
    for(int arg = 0; arg < argc; arg++){ 
        std::stringstream buffer;
        if(strcmp(argv[arg], "-error") == 0){
            buffer << argv[arg+1];
            buffer >> eps;
        }
        else if(strcmp(argv[arg], "-iter") == 0){
            buffer << argv[arg+1];
            buffer >> iter_max;
        }
        else if(strcmp(argv[arg], "-size") == 0){
            buffer << argv[arg+1];
            buffer >> size;
        }
    }

	size_t totalSize = size * size;

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Выбор видеокарт
    int DEVICE, COUNT_DEVICE;
    MPI_Init(&argc, &argv);

    MPI_Comm_rank(MPI_COMM_WORLD, &DEVICE);
    MPI_Comm_size(MPI_COMM_WORLD, &COUNT_DEVICE);

    hipSetDevice(DEVICE);

    if (DEVICE == 0)
        std::cout << "Settings: " << "\n\tMin error: " << eps << "\n\tMax iteration: " << iter_max << "\n\tSize: " << size << "x" << size << std::endl;

	if (DEVICE!=0)
        hipDeviceEnablePeerAccess(DEVICE - 1, 0);
    if (DEVICE!=COUNT_DEVICE-1)
        hipDeviceEnablePeerAccess(DEVICE + 1, 0);

	size_t size_y = size / COUNT_DEVICE + 1;
    if (DEVICE != COUNT_DEVICE - 1 && DEVICE != 0) size_y += 1;
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Выделения памяти
	double *A, *A_Device, *A_new_Device, *A_error_Device, *deviceError, *tempStorage = NULL;
    size_t tempStorageSize = 0;

    hipHostMalloc(&A, sizeof(double) * totalSize);

    for (int j = 0; j < size; j++)  {
        A[j] = 10.0 + j * 10.0 / (size - 1);
        A[j * size] = 10.0 + j * 10.0 / (size - 1);
        A[size - 1 + j * size] = 20.0 + j * 10.0 / (size - 1);
        A[size * (size - 1) + j] = 20.0 + j * 10.0 / (size - 1);
    }

    dim3 threads(THREAD);
    dim3 blocks(size/THREAD, size_y);

    printf("%d: %d %d %d %d\n", DEVICE, threads.x, threads.y, blocks.x, blocks.y);

	hipMalloc(&A_Device, sizeof(double) * size * size_y);
	hipMalloc(&A_new_Device, sizeof(double) * size * size_y);
	hipMalloc(&A_error_Device, sizeof(double) * size * size_y);
	hipMalloc(&deviceError, sizeof(double));


	size_t offset = (DEVICE != 0) ? size : 0;
 	hipMemcpy(A_Device, A + (size * size * DEVICE / COUNT_DEVICE) - offset, sizeof(double) * size * size_y, hipMemcpyHostToDevice);
	hipMemcpy(A_new_Device, A + (size * size * DEVICE / COUNT_DEVICE) - offset, sizeof(double) * size * size_y, hipMemcpyHostToDevice);

	hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, A_error_Device, deviceError, size * size_y);
	hipMalloc(&tempStorage, tempStorageSize);
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Основной цикл
    size_t iter = 0;
    double error = 1.0;
	clock_t begin = clock();
	while((iter < iter_max) && error > eps)	{
		iterate<<<blocks, threads>>>(A_Device, A_new_Device, size, size_y);
		iter++;
		// Расчитываем ошибку каждую сотую итерацию
		if (iter % UPDATE == 0) {
            subtraction<<<blocks, threads>>>(A_new_Device, A_Device, A_error_Device, size);
			hipcub::DeviceReduce::Max(tempStorage, tempStorageSize, A_error_Device, deviceError, size * size_y);
			hipMemcpyAsync(&error, deviceError, sizeof(double), hipMemcpyDeviceToHost);

			// Находим максимальную ошибку среди всех и передаём её всем процессам
			MPI_Allreduce((void*)&error,(void*)&error, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);
		}

		if (DEVICE != 0)                // Обмен верхней границей
            MPI_Sendrecv(A_new_Device + size + 1, size - 2, MPI_DOUBLE, DEVICE - 1, 0, A_new_Device + 1, size - 2, MPI_DOUBLE, DEVICE - 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
		
        if (DEVICE != COUNT_DEVICE - 1) // Обмен нижней границей
            MPI_Sendrecv(A_new_Device + (size_y - 2) * size + 1, size - 2, MPI_DOUBLE, DEVICE + 1, 0, A_new_Device + (size_y - 1) * size + 1, size - 2, MPI_DOUBLE, DEVICE + 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);

		std::swap(A_Device, A_new_Device);
	}

	clock_t end = clock();
	if (DEVICE == 0) {
		std::cout << "Result:\n\tIter: " << iter << "\n\tError: " << error << "\n\tTime: " << 1.0 * (end - begin) / CLOCKS_PER_SEC << std::endl;
	}
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////// Чистка памяти
	hipFree(A_Device);
	hipFree(A_new_Device);
	hipFree(A_error_Device);
	hipFree(tempStorage);

	MPI_Finalize();

	return 0;
}