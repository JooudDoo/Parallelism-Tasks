#include "hip/hip_runtime.h"
#include "TcuBlas.cuh"

#include <random>
#include <stdexcept>

FullyConnectedLayer::FullyConnectedLayer(){
    throw std::runtime_error("You should specifed in_size and out_size");
}

FullyConnectedLayer::FullyConnectedLayer(int input_size, int output_size) : input_size_(input_size), output_size_(output_size){
    hipMalloc(&d_input_, input_size_ * sizeof(calc_type));
    hipMalloc(&d_output_, output_size_ * sizeof(calc_type));
    hipMalloc(&d_weight_, input_size_ * output_size_ * sizeof(calc_type));
    hipMalloc(&d_bias_, output_size_ * sizeof(calc_type));
    hipblasCreate(&handle_);
}

FullyConnectedLayer::~FullyConnectedLayer() {
    hipblasDestroy(handle_);
    hipFree(d_input_);
    hipFree(d_output_);
    hipFree(d_weight_);
    hipFree(d_bias_);
}

calc_type* FullyConnectedLayer::forward(const calc_type* input) {
    hipblasSetMatrix(input_size_, 1, sizeof(calc_type), input, input_size_, d_input_, input_size_);
    hipblasSgemv(handle_, HIPBLAS_OP_T, input_size_, output_size_, &alpha_, d_weight_, input_size_, d_input_, 1, &beta_, d_output_, 1);
    hipblasSgeam(handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, output_size_, input_size_, &alpha_, d_bias_, output_size_, &alpha_, d_output_, output_size_, d_output_, output_size_);
    // hipblasSaxpy(handle_, output_size_, &alpha_, d_bias_, 1, d_output_, 1);
    return d_output_;
}

void FullyConnectedLayer::set_weights(const PARAMETR_SOURCE source_type, const std::string file_path){
    if(source_type == PARAMETR_SOURCE::FILE){
        load_parameter(d_weight_, file_path, {input_size_, output_size_});
    }
    else if(source_type == PARAMETR_SOURCE::GENERATE){
        generate_parmeter(d_weight_, {input_size_, output_size_});
    }
}

void FullyConnectedLayer::set_bias(const PARAMETR_SOURCE source_type, const std::string file_path){
    if(source_type == PARAMETR_SOURCE::FILE){
        load_parameter(d_bias_, file_path, {1, output_size_});
    }
    else if(source_type == PARAMETR_SOURCE::GENERATE){
        generate_parmeter(d_bias_, {1, output_size_});
    }
}

void FullyConnectedLayer::generate_parmeter(calc_type* target, const std::pair<int, int> size, const int random_seed){
    int in_size = size.first;
    int out_size = size.second;
    
    std::default_random_engine rangen;
    if(random_seed != -1){
        rangen.seed(random_seed);
    }

    calc_type disp = std::sqrt((double)2/out_size);
    calc_type mean = 0;

    std::normal_distribution<calc_type> distribution(mean, disp);

    if(target == nullptr){
        hipMalloc(&target, in_size*out_size*sizeof(calc_type));
    }

    calc_type* temp = (calc_type*)malloc(in_size*out_size*sizeof(calc_type));

    for(size_t i = 0; i < in_size*out_size; i++){
        temp[i] = distribution(rangen);
    }

    hipMemcpy(target, temp, in_size*out_size*sizeof(calc_type), hipMemcpyHostToDevice);

    free(temp);
}